#include "hip/hip_runtime.h"
#include <optix_device.h>
#include "LaunchParams.h"
#include <vec_math.h>

/* Compile with:
nvcc.exe -O3 -use_fast_math -arch=compute_30 -code=sm_30 -I "C:\ProgramData\NVIDIA Corporation\OptiX SDK 7.0.0\include" -I "C:\Program Files (x86)\Microsoft Visual Studio\2019\Community\VC\Tools\MSVC\14.24.28314\include" -I "." -m 64 -ptx -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\Community\VC\Tools\MSVC\14.24.28314\bin\Hostx64\x64" raytracing.cu -o raytracing.ptx
*/

/*! launch parameters in constant memory, filled in by optix upon
    optixLaunch (this gets filled in from the buffer we pass to
    optixLaunch) */
extern "C" {
    __constant__ LaunchParams optixLaunchParams;
}

// for this simple example, we have a single ray type
enum { PHONG_RAY_TYPE = 0, SHADOW_RAY_TYPE,  RAY_TYPE_COUNT };

//closest hit radiance
extern "C" __global__ void __closesthit__radiance() {
    
    const TriangleMeshSBTData &sbtData = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // gather basic info
    const int primID = optixGetPrimitiveIndex();
    const uint3 index = sbtData.index[primID];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;


    // compute triangle normal using either shading normal or gnormal as fallback:
    const float3 &A = make_float3(sbtData.vertexD.position[index.x]);
    const float3 &B = make_float3(sbtData.vertexD.position[index.y]);
    const float3 &C = make_float3(sbtData.vertexD.position[index.z]);

    float3 Ns;
    float3 Ng = cross(B-A,C-A);
    if(sbtData.vertexD.normal) 
        Ns = make_float3((1.f-u-v) * sbtData.vertexD.normal[index.x] + u * sbtData.vertexD.normal[index.y] + v * sbtData.vertexD.normal[index.z]);
    else 
        Ns = Ng;
    
    // Face forward + Normalization
    const float3 rayDir = optixGetWorldRayDirection();
    
    if (dot(rayDir,Ng) > 0.f) Ng = -Ng;
    Ng = normalize(Ng);
    
    if (dot(Ng,Ns) < 0.f) Ns -= 2.f*dot(Ng,Ns)*Ng;
    Ns = normalize(Ns);

    // Lambert Diffuse
    float3 diffuseColor = sbtData.color;
    if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {
      const float2 tc = make_float2((1.f-u-v) * sbtData.vertexD.texCoord0[index.x] + u * sbtData.vertexD.texCoord0[index.y] + v * sbtData.vertexD.texCoord0[index.z]);
      float4 fromTexture = tex2D<float4>(sbtData.texture,tc.x,tc.y);
      diffuseColor *= make_float3(fromTexture);
    }

    // Shadow
    const float3 surfPos = make_float3((1.f-u-v) * sbtData.vertexD.position[index.x] + u * sbtData.vertexD.position[index.y] + v * sbtData.vertexD.position[index.z]);
    const float3 lightPos = make_float3(-907.108f, 2205.875f, -400.0267f);
    const float3 lightDir = lightPos - surfPos;

    // Trace Shadow Ray
    float3 lightVisibility = make_float3(0.f,0.f,0.f);

    uint32_t u0, u1;
    packPointer( &lightVisibility, u0, u1 );
    optixTrace(optixLaunchParams.traversable,
               surfPos + 1e-3f * Ng,
               lightDir,
               1e-3f,      // tmin
               1.f-1e-3f,  // tmax
               0.0f,       // rayTime
               OptixVisibilityMask( 255 ),
               OPTIX_RAY_FLAG_DISABLE_ANYHIT
               | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT
               | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
               SHADOW_RAY_TYPE,            // SBT offset
               RAY_TYPE_COUNT,               // SBT stride
               SHADOW_RAY_TYPE,            // missSBTIndex 
               u0, u1 );

    // Final shading: ambient, directional ambient and shadowing
    const float cosDN = 0.1f + .8f*fabsf(dot(rayDir,Ns));
    float3 &prd = *(float3*)getPRD<float3>();
    prd = (.1f + (.2f + .8f*lightVisibility) * cosDN) * diffuseColor;
    
}

//any hit radiance
extern "C" __global__ void __anyhit__radiance() {

}

//miss radiance
extern "C" __global__ void __miss__radiance() {

}

//closest hit shadow
extern "C" __global__ void __closesthit__shadow() {

}

//any hit shadow
extern "C" __global__ void __anyhit__shadow() {

}

//miss shadow
extern "C" __global__ void __miss__shadow() {

}

//closest hit radiance para grades
extern "C" __global__ void __closesthit__radiance_grade() {

}

//any hit radiance para grades
extern "C" __global__ void __anyhit__radiance_grade() {

}

//miss radiance para grades
extern "C" __global__ void __miss__radiance_grade() {
    
}

//closest hit shadow para grades
extern "C" __global__ void __closesthit__shadow_grade() {

}

//any hit shadow para grades
extern "C" __global__ void __anyhit__shadow_grade() {

}

//miss shadow para grades
extern "C" __global__ void __miss__shadow_grade() {

}

//closest hit radiance para vidros
extern "C" __global__ void __closesthit__radiance_vidro() {

}

//any hit radiance para vidros
extern "C" __global__ void __anyhit__radiance_vidro() {

}

//miss radiance para vidros
extern "C" __global__ void __miss__radiance_vidro() {
    
}

//closest hit shadow para vidros
extern "C" __global__ void __closesthit__shadow_vidro() {

}

//any hit shadow para vidros
extern "C" __global__ void __anyhit__shadow_vidro() {

}

//miss shadow para vidros
extern "C" __global__ void __miss__shadow_vidro() {

}

//Ray Deployment
extern "C" __global__ void __raygen__renderFrame() {

    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const auto &camera = optixLaunchParams.camera;
    // ray payload
    float3 pixelColorPRD = make_float3(1.f);
    uint32_t u0, u1;
    packPointer( &pixelColorPRD, u0, u1 );
    // compute ray direction
    // normalized screen plane position, in [-1, 1]^2
    const float2 screen(make_float2(ix+.5f,iy+.5f)
                        / make_float2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y) * 2.0 - 1.0);
    // note: nau already takes into account the field of view when computing
    // camera horizontal and vertical
    float3 rayDir = normalize(camera.direction
                              + screen.x * camera.horizontal
                              + screen.y * camera.vertical);

    // trace primary ray
    optixTrace(optixLaunchParams.traversable,
        camera.position,
        rayDir,
        0.f, // tmin
        1e20f, // tmax
        0.0f, // rayTime
        OptixVisibilityMask( 255 ),
        OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
        PHONG_RAY_TYPE, // SBT offset
        RAY_TYPE_COUNT, // SBT stride
        PHONG_RAY_TYPE, // missSBTIndex
        u0, u1 );
    
    //convert float (0-1) to int (0-255)
    const int r = int(255.0f*pixelColorPRD.x);
    const int g = int(255.0f*pixelColorPRD.y);
    const int b = int(255.0f*pixelColorPRD.z);
    // convert to 32-bit rgba value
    const uint32_t rgba = 0xff000000
    | (r<<0) | (g<<8) | (b<<16);
    // compute index
    const uint32_t fbIndex = ix+iy*optixGetLaunchDimensions().x;
    // write to output buffer
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
    
    if(optixLaunchParams.frame.frame == 0 && ix == 0 && iy == 0){
        // print info to console
        printf("===========================================\n");
        printf("Nau Ray-Tracing Hello World\n");
        printf("Launch size: %i x %i\n", ix, iy);
        printf("Camera Direction: %f %f %f\n",
        optixLaunchParams.camera.direction.x,
        optixLaunchParams.camera.direction.y,
        optixLaunchParams.camera.direction.z);
        printf("===========================================\n");
    }

}

